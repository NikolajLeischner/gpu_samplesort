#include "gpu_dispatching.h"
#include "distributions.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "../samplesort/samplesort.h"


namespace Benchmark {
    template<typename KeyType>
    void sort_by_algorithm(Algorithm::Value algorithm, bool keys_only, std::vector<KeyType> &data) {
        KeyType *device_keys(0);
        std::uint64_t *device_values(0);
        size_t size = sizeof(KeyType) * data.size();
        hipMalloc((void **) &device_keys, size);
        hipMemcpy(device_keys, data.data(), size, hipMemcpyHostToDevice);

        if (keys_only) {
            if (algorithm == Algorithm::Value::thrust) {
                thrust::device_ptr <KeyType> keys_ptr(device_keys);
                thrust::sort(keys_ptr, keys_ptr + data.size());
            } else if (algorithm == Algorithm::Value::samplesort) {
                SampleSort::sort(device_keys, device_keys + data.size());
            }
        } else {
            auto values = Distributions::uniform<std::uint64_t>(data.size(), Distributions::Settings(64, 1));

            hipMalloc((void **) &device_values, values.memory_size());
            hipMemcpy(device_values, values.begin(), values.memory_size(), hipMemcpyHostToDevice);

            if (algorithm == Algorithm::Value::thrust) {
                thrust::device_ptr <KeyType> keys_ptr(device_keys);
                thrust::device_ptr <std::uint64_t> values_ptr(device_values);
                thrust::sort_by_key(keys_ptr, keys_ptr + data.size(), values_ptr);
            } else if (algorithm == Algorithm::Value::samplesort) {
                SampleSort::sort_by_key(device_keys, device_keys + data.size(), device_values);
            }

            hipMemcpy(values.as_vector().data(), device_values, values.memory_size(), hipMemcpyDeviceToHost);
        }

        hipMemcpy(data.data(), device_keys, size, hipMemcpyDeviceToHost);
        hipFree(device_keys);
        hipFree(device_values);
    }


    template void sort_by_algorithm(Algorithm::Value algorithm, bool keys_only,
                                        std::vector<std::uint16_t> &data);

    template void sort_by_algorithm(Algorithm::Value algorithm, bool keys_only,
                                        std::vector<std::uint32_t> &data);

    template void sort_by_algorithm(Algorithm::Value algorithm, bool keys_only,
                                        std::vector<std::uint64_t> &data);
}