#include "gpu_dispatching.h"
#include "distributions.h"

namespace Benchmark {
    template<typename KeyType>
    double benchmark_algorithm(Benchmark::Algorithm::Value algorithm, bool keys_have_values, std::vector<KeyType> &data) {

        KeyType *device_keys(0);
        std::uint64_t *device_values(0);
        size_t size = sizeof(KeyType) * data.size();
        hipMalloc((void **) &device_keys, size);
        hipMemcpy(device_keys, data.data(), size, hipMemcpyHostToDevice);


        if (keys_have_values) {

            auto values = Distributions::uniform<std::uint64_t>(data.size(), Distributions::Settings(64, 1));

            hipMalloc((void **) &device_values, values.memory_size());
            hipMemcpy(device_values, values.begin(), values.memory_size(), hipMemcpyHostToDevice);

            hipMemcpy(values.as_vector().data(), device_values, values.memory_size(), hipMemcpyDeviceToHost);
        }

        hipMemcpy(data.data(), device_keys, size, hipMemcpyDeviceToHost);
        hipFree(device_keys);
        hipFree(device_values);

        return 0.0;
    }

    template double benchmark_algorithm(Benchmark::Algorithm::Value algorithm, bool keys_have_values,
                                        std::vector <std::uint16_t> &data);

    template double benchmark_algorithm(Benchmark::Algorithm::Value algorithm, bool keys_have_values,
                                        std::vector <std::uint32_t> &data);

    template double benchmark_algorithm(Benchmark::Algorithm::Value algorithm, bool keys_have_values,
                                        std::vector <std::uint64_t> &data);
}